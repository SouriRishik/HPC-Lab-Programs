#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void transformMatrix(int *mat, int M, int N) {
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    if (r < M && c < N) {
        mat[r * N + c] = (int)powf(mat[r * N + c], r + 1);
    }
}

int main() {
    int M, N;
    scanf("%d%d", &M, &N);
    int *h_mat = (int*)malloc(M * N * sizeof(int));
    for (int i = 0; i < M * N; i++) scanf("%d", &h_mat[i]);

    int *d_mat;
    hipMalloc(&d_mat, M * N * sizeof(int));
    hipMemcpy(d_mat, h_mat, M * N * sizeof(int), hipMemcpyHostToDevice);

    dim3 block(16,16), grid((N + 15)/16, (M + 15)/16);
    transformMatrix<<<grid, block>>>(d_mat, M, N);
    hipMemcpy(h_mat, d_mat, M * N * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) printf("%d ", h_mat[i * N + j]);
        printf("\n");
    }

    free(h_mat);
    hipFree(d_mat);
    return 0;
}