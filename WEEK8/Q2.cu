
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

__global__ void reverseWords(char *str, int *word_starts, int *word_ends, int N)
{
    int idx = threadIdx.x;
    if (idx < N)
    {
        int start = word_starts[idx];
        int end = word_ends[idx];
        while (start < end)
        {
            char temp = str[start];
            str[start++] = str[end--];
            str[end] = temp;
        }
    }
}

int main()
{
    char str[256];
    printf("Enter a string: ");
    scanf(" %[^\n]", str);

    int word_starts[50], word_ends[50], N = 0;
    int len = strlen(str);
    int i = 0;
    while (i < len)
    {
        while (i < len && str[i] == ' ')
            i++;
        if (i < len)
            word_starts[N] = i;
        while (i < len && str[i] != ' ')
            i++;
        if (i > 0 && str[i - 1] != ' ')
            word_ends[N] = i - 1;
        if (i <= len)
            N++;
    }

    char *d_str;
    int *d_starts, *d_ends;
    hipMalloc(&d_str, len);
    hipMalloc(&d_starts, N * sizeof(int));
    hipMalloc(&d_ends, N * sizeof(int));

    hipMemcpy(d_str, str, len, hipMemcpyHostToDevice);
    hipMemcpy(d_starts, word_starts, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_ends, word_ends, N * sizeof(int), hipMemcpyHostToDevice);

    reverseWords<<<1, N>>>(d_str, d_starts, d_ends, N);

    hipMemcpy(str, d_str, len, hipMemcpyDeviceToHost);

    printf("Output: %s\n", str);

    hipFree(d_str);
    hipFree(d_starts);
    hipFree(d_ends);
    return 0;
}