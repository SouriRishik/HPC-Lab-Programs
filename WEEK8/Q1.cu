
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

__global__ void repeatString(char *S, char *out, int len, int N)
{
    int idx = threadIdx.x;
    if (idx < N)
    {
        for (int i = 0; i < len; i++)
        {
            out[idx * len + i] = S[i];
        }
    }
}

int main()
{
    char S[100];
    int N;
    printf("Enter string: ");
    scanf("%s", S);
    printf("Enter N: ");
    scanf("%d", &N);

    int len = strlen(S);
    int outLen = len * N;
    char *d_S, *d_out;
    char *out = (char *)malloc(outLen + 1);

    hipMalloc(&d_S, len);
    hipMalloc(&d_out, outLen);

    hipMemcpy(d_S, S, len, hipMemcpyHostToDevice);

    repeatString<<<1, N>>>(d_S, d_out, len, N);

    hipMemcpy(out, d_out, outLen, hipMemcpyDeviceToHost);

    out[outLen] = '\0';
    printf("Output String: %s\n", out);

    hipFree(d_S);
    hipFree(d_out);
    free(out);
    return 0;
}